#include "hip/hip_runtime.h"
#pragma once

#include "modules/reconstruction/Projector.h"
#include "modules/reconstruction/CudaErrorHandler.h"

#include <vtkTIFFWriter.h>

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#define _DEBUGPROJECTION
#define _DEBUGPBACKROJECTION
#define LOGPROJECTION
#define LOGPROJECTIONINITERATIONS_


__forceinline __device__ bool floatEq( const float p_A, const float p_B, const float p_tolerance = 0.0000001F )
{
    if( p_tolerance < 0 )
    {
        return ( p_A - p_B ) < -p_tolerance && ( p_A - p_B ) > p_tolerance;
    }
    else
    {
        return ( p_A - p_B ) < p_tolerance && ( p_A - p_B ) > -p_tolerance;
    }
}

__forceinline __device__ bool floatSupEq( const float p_A, const float p_B, const float p_tolerance = 0.0000001F )
{
    return p_A > p_B || floatEq( p_A, p_B, p_tolerance );
}

__forceinline __device__ bool floatInfEq( const float p_A, const float p_B, const float p_tolerance = 0.0000001F )
{
    return p_A < p_B || floatEq( p_A, p_B, p_tolerance );
}

// Important: user of this function: you are responsible of the cuda free of p_alphas pointer
__forceinline __device__ int FindProjectionAlphas_d( const float3 p_rayDirectorVector, const float3 p_currentSourcePosition, const dim3 p_volumeDimension, const int p_maxNumberOfDimensions, const float p_floatTolerance, float * p_alphas )
{
    // Volume half length in floating voxel reference
    float3 volumeHalfLength;
    volumeHalfLength.x = static_cast<float>( p_volumeDimension.x ) / 2.F;
    volumeHalfLength.y = static_cast<float>( p_volumeDimension.y ) / 2.F;
    volumeHalfLength.z = static_cast<float>( p_volumeDimension.z ) / 2.F;

    // Calculate max and min alpha- and i-values.
    float3 alpha1;    // alpha corresponding to the bottom left front vertex
    alpha1.x = fabs( p_rayDirectorVector.x ) > p_floatTolerance ? ( -volumeHalfLength.x - p_currentSourcePosition.x ) / p_rayDirectorVector.x : 0.F;
    alpha1.y = fabs( p_rayDirectorVector.y ) > p_floatTolerance ? ( -volumeHalfLength.y - p_currentSourcePosition.y ) / p_rayDirectorVector.y : 0.F;
    alpha1.z = fabs( p_rayDirectorVector.z ) > p_floatTolerance ? ( -volumeHalfLength.z - p_currentSourcePosition.z ) / p_rayDirectorVector.z : 0.F;
    float3 alphaN;    // alpha corresponding to the top right back vertex
    alphaN.x = fabs( p_rayDirectorVector.x ) > p_floatTolerance ? ( volumeHalfLength.x - p_currentSourcePosition.x ) / p_rayDirectorVector.x : 0.F;
    alphaN.y = fabs( p_rayDirectorVector.y ) > p_floatTolerance ? ( volumeHalfLength.y - p_currentSourcePosition.y ) / p_rayDirectorVector.y : 0.F;
    alphaN.z = fabs( p_rayDirectorVector.z ) > p_floatTolerance ? ( volumeHalfLength.z - p_currentSourcePosition.z ) / p_rayDirectorVector.z : 0.F;


    float3 alphaMins;    // alpha corresponding to the bottom left front vertex
    alphaMins.x = fmin( alpha1.x, alphaN.x );
    alphaMins.y = fmin( alpha1.y, alphaN.y );
    alphaMins.z = fmin( alpha1.z, alphaN.z );
    float3 alphaMaxs;    // alpha corresponding to the top right back vertex
    alphaMaxs.x = fmax( alpha1.x, alphaN.x );
    alphaMaxs.y = fmax( alpha1.y, alphaN.y );
    alphaMaxs.z = fmax( alpha1.z, alphaN.z );


    // We take the highest of the mins to catch the incoming intersection alpha
    auto alphaMin = fmax( fmax( 0.F, alphaMins.x ), fmax( alphaMins.y, alphaMins.z ) );
    // We take the lowest of the maxs to catch the outgoing intersection alpha
    auto alphaMax = fmin( fmin( 1.F, alphaMaxs.x ), fmin( alphaMaxs.y, alphaMaxs.z ) );

    if( floatSupEq( alphaMin, alphaMax ) )
    {
#ifdef DEBUGPROJECTION
        printf( "Id[%d] alphaMin = %f is greater or equal to alphaMax = %f !!! \n", globalProjectionIndex, alphaMin, alphaMax );
#endif
        return;
    }

    // min and max indices in each direction
    dim3 minIndices, maxIndices;
    if( p_rayDirectorVector.x > 0 )
    {
        maxIndices.x = max( 0, min( static_cast<int>( p_volumeDimension.x ) - 1, static_cast<int>( volumeHalfLength.x + p_currentSourcePosition.x + alphaMax * p_rayDirectorVector.x ) ) );
        minIndices.x = max( 0, min( static_cast<int>( maxIndices.x ), static_cast<int>( p_volumeDimension.x ) - static_cast<int>( volumeHalfLength.x - p_currentSourcePosition.x - alphaMin * p_rayDirectorVector.x ) ) );
    }
    else
    {
        maxIndices.x = max( 0, min( static_cast<int>( p_volumeDimension.x ) - 1, static_cast<int>( volumeHalfLength.x + p_currentSourcePosition.x + alphaMin * p_rayDirectorVector.x ) ) );
        minIndices.x = max( 0, min( static_cast<int>( maxIndices.x ), static_cast<int>( p_volumeDimension.x ) - static_cast<int>( volumeHalfLength.x - p_currentSourcePosition.x - alphaMax * p_rayDirectorVector.x ) ) );
    }

    if( p_rayDirectorVector.y > 0 )
    {
        maxIndices.y = max( 0, min( static_cast<int>( p_volumeDimension.y ) - 1, static_cast<int>( volumeHalfLength.y + p_currentSourcePosition.y + alphaMax * p_rayDirectorVector.y ) ) );
        minIndices.y = max( 0, min( static_cast<int>( maxIndices.y ), static_cast<int>( p_volumeDimension.y ) - static_cast<int>( volumeHalfLength.y - p_currentSourcePosition.y - alphaMin * p_rayDirectorVector.y ) ) );
    }
    else
    {
        maxIndices.y = max( 0, min( static_cast<int>( p_volumeDimension.y ) - 1, static_cast<int>( volumeHalfLength.y + p_currentSourcePosition.y + alphaMin * p_rayDirectorVector.y ) ) );
        minIndices.y = max( 0, min( static_cast<int>( maxIndices.y ), static_cast<int>( p_volumeDimension.y ) - static_cast<int>( volumeHalfLength.y - p_currentSourcePosition.y - alphaMax * p_rayDirectorVector.y ) ) );
    }

    if( p_rayDirectorVector.z > 0 )
    {
        maxIndices.z = max( 0, min( static_cast<int>( p_volumeDimension.z ) - 1, static_cast<int>( volumeHalfLength.z + p_currentSourcePosition.z + alphaMax * p_rayDirectorVector.z ) ) );
        minIndices.z = max( 0, min( static_cast<int>( maxIndices.z ), static_cast<int>( p_volumeDimension.z ) - static_cast<int>( volumeHalfLength.z - p_currentSourcePosition.z - alphaMin * p_rayDirectorVector.z ) ) );
    }
    else
    {
        maxIndices.z = max( 0, min( static_cast<int>( p_volumeDimension.z ) - 1, static_cast<int>( volumeHalfLength.z + p_currentSourcePosition.z + alphaMin * p_rayDirectorVector.z ) ) );
        minIndices.z = max( 0, min( static_cast<int>( maxIndices.z ), static_cast<int>( p_volumeDimension.z ) - static_cast<int>( volumeHalfLength.z - p_currentSourcePosition.z - alphaMax * p_rayDirectorVector.z ) ) );
    }

    // Calculate alpha-sets for x, y and z.
    auto globalSize = ( maxIndices.x - minIndices.x )
                      + ( maxIndices.y - minIndices.y )
                      + ( maxIndices.z - minIndices.z )
                      + 3;    // each component goes from minIndices to maxIndices (included) than
    // it includes (maxIndices - minIndices + 1) elements

    if( globalSize > p_maxNumberOfDimensions )
    {
        // TODO : DIMENSION PROBLEM TO HANDLE
    }

    for( auto i = 0; i < globalSize; i++ )
    {
        p_alphas[i] = 1.0;
    }
    auto initialIndex = 0;
    auto finalComponentWiseIndex = initialIndex + ( maxIndices.x - minIndices.x ) + 1;

    // x intersections findings
    if( fabs( p_rayDirectorVector.x ) > p_floatTolerance )
    {
        const auto xAlphaDiff = 1.F / p_rayDirectorVector.x;
        if( p_rayDirectorVector.x > 0 )
        {
            p_alphas[initialIndex] = xAlphaDiff * ( -volumeHalfLength.x - p_currentSourcePosition.x + static_cast<float>( minIndices.x ) );
        }
        else
        {
            p_alphas[initialIndex] = xAlphaDiff * ( -volumeHalfLength.x - p_currentSourcePosition.x + static_cast<float>( maxIndices.x ) );
        }

        for( auto index = initialIndex + 1; index < finalComponentWiseIndex; index++ )
        {
            p_alphas[index] = p_alphas[index - 1] + fabs( xAlphaDiff );
        }
    }

    initialIndex = finalComponentWiseIndex;
    finalComponentWiseIndex += ( maxIndices.y - minIndices.y ) + 1;
    // y intersections findings
    if( fabs( p_rayDirectorVector.y ) > p_floatTolerance )
    {
        const auto yAlphaDiff = 1.F / p_rayDirectorVector.y;
        if( p_rayDirectorVector.y > 0 )
        {
            p_alphas[initialIndex] = yAlphaDiff * ( -volumeHalfLength.y - p_currentSourcePosition.y + static_cast<float>( minIndices.y ) );
        }
        else
        {
            p_alphas[initialIndex] = yAlphaDiff * ( -volumeHalfLength.y - p_currentSourcePosition.y + static_cast<float>( maxIndices.y ) );
        }

        for( auto index = initialIndex + 1; index < finalComponentWiseIndex; index++ )
        {
            p_alphas[index] = p_alphas[index - 1] + fabs( yAlphaDiff );
        }
    }

    initialIndex = finalComponentWiseIndex;
    finalComponentWiseIndex += ( maxIndices.z - minIndices.z ) - 1;

    // z intersections findings
    if( fabs( p_rayDirectorVector.z ) > p_floatTolerance )
    {
        const auto zAlphaDiff = 1.F / p_rayDirectorVector.z;
        if( p_rayDirectorVector.z > 0 )
        {
            p_alphas[initialIndex] = zAlphaDiff * ( -volumeHalfLength.z - p_currentSourcePosition.z + static_cast<float>( minIndices.z ) );
        }
        else
        {
            p_alphas[initialIndex] = zAlphaDiff * ( -volumeHalfLength.z - p_currentSourcePosition.z + static_cast<float>( maxIndices.z ) );
        }

        for( auto index = initialIndex + 1; index < finalComponentWiseIndex; index++ )
        {
            p_alphas[index] = p_alphas[index - 1] + fabs( zAlphaDiff );
        }
    }


    // SELECTION SORT
    for( auto i = 0; i < globalSize - 1; i++ )
    {
        // Find the minimum element in unsorted array
        auto min_idx = i;
        for( auto j = i + 1; j < globalSize; j++ )
        {
            if( p_alphas[j] < p_alphas[min_idx] )
            {
                min_idx = j;
            }
        }
        // Swap the found minimum element with the first element
        auto tmpFloat = p_alphas[min_idx];
        p_alphas[min_idx] = p_alphas[i];
        p_alphas[i] = tmpFloat;
    }

    return globalSize;
}

// For this procedure, the coordinate system is the one centered in C, the pave volume
// center.
__global__ void CudaPerformProjection(
  const float * p_volumeBuffer,
  const dim3 p_volumeDimension,
  const float3 p_volumeOriginInWorld,
  const float3 p_volumeVoxelsSpacing,
  float * p_projectionsBuffer,
  const dim3 p_projectionsDimension,
  const float2 p_projectionsPixelsSpacing,
  const float3 * p_projectionsOriginInWorld,
  const float3 * p_sourcesPositions )
{
    const auto nbProjections = p_projectionsDimension.z;
    const auto nPixelsInOneRoi = p_projectionsDimension.x * p_projectionsDimension.y;
    const auto nProjectionsPixels = nbProjections * nPixelsInOneRoi;
    const auto projectionIndex = blockIdx.z * blockDim.z + threadIdx.z;
    const auto xProjPixel = blockIdx.x * blockDim.x + threadIdx.x;
    const auto yProjPixel = blockIdx.y * blockDim.y + threadIdx.y;

    constexpr int maxNumberOfDimensions = 2000;
    constexpr auto floatTolerance = 0.000001F;

#ifdef DEBUGPROJECTION
    // debugging index for some useful logging
    const auto debuggingProjIndex = 6;
    const auto debuggingProjX = 23;
    const auto debuggingProjY = 15;
    const auto debuggingIndex = debuggingProjIndex * nPixelsInOneRoi + debuggingProjY * p_projectionsDimension.x + debuggingProjX;
#endif


    if( projectionIndex < 0 || projectionIndex >= nbProjections
        || xProjPixel < 0 || xProjPixel >= p_projectionsDimension.x
        || yProjPixel < 0 || yProjPixel >= p_projectionsDimension.y )
    {
        return;
    }

    const auto globalProjectionIndex = projectionIndex * nPixelsInOneRoi + yProjPixel * p_projectionsDimension.x + xProjPixel;

    if( globalProjectionIndex < 0 || globalProjectionIndex >= nProjectionsPixels )
    {
#ifdef DEBUGPROJECTION
        printf( "Id[%d] NOT POSSIBLE TO BE THERE !!! \n", globalProjectionIndex );
#endif
        return;
    }
    p_projectionsBuffer[globalProjectionIndex] = 0.F;

#ifdef DEBUGPROJECTION
    if( globalProjectionIndex == debuggingIndex )
    {
        printf( "___basic computations___\n Id[%d] p_projectionsDimension.x : %d\n p_projectionsDimension.y : %d\n nbProjections : %d\n nPixelsInOneRoi : %d\n nProjectionsPixels : %d\n blockIdx: %d , %d , %d\n blockDim: %d , %d , %d\n threadIdx: %d , %d , %d\n projectionIndex : %d\n xProjPixel : %d\n yProjPixel : %d\n_________\n",
                globalProjectionIndex,
                p_projectionsDimension.x,
                p_projectionsDimension.y,
                nbProjections,
                nPixelsInOneRoi,
                nProjectionsPixels,
                blockIdx.x,
                blockIdx.y,
                blockIdx.z,
                blockDim.x,
                blockDim.y,
                blockDim.z,
                threadIdx.x,
                threadIdx.y,
                threadIdx.z,
                projectionIndex,
                xProjPixel,
                yProjPixel );
    }
#endif


    // source positions (in the floating voxel coordinate system with same origin as world)
    // origin is the volume center
    float3 currentSourcePositionFloatingVoxel;
    currentSourcePositionFloatingVoxel.x = p_sourcesPositions[projectionIndex].x / p_volumeVoxelsSpacing.x;
    currentSourcePositionFloatingVoxel.y = p_sourcesPositions[projectionIndex].y / p_volumeVoxelsSpacing.y;
    currentSourcePositionFloatingVoxel.z = p_sourcesPositions[projectionIndex].z / p_volumeVoxelsSpacing.z;

    float3 currentProjectionPositionFloatingVoxel;

    currentProjectionPositionFloatingVoxel.x = ( p_projectionsOriginInWorld[projectionIndex].x + ( static_cast<float>( xProjPixel ) + 0.5F ) * p_projectionsPixelsSpacing.x ) / p_volumeVoxelsSpacing.x;
    currentProjectionPositionFloatingVoxel.y = ( p_projectionsOriginInWorld[projectionIndex].y + ( static_cast<float>( yProjPixel ) + 0.5F ) * p_projectionsPixelsSpacing.y ) / p_volumeVoxelsSpacing.y;
    currentProjectionPositionFloatingVoxel.z = p_projectionsOriginInWorld[projectionIndex].z / p_volumeVoxelsSpacing.z;

    float3 rayDirectorVectorFloatingVoxel;
    rayDirectorVectorFloatingVoxel.x = currentProjectionPositionFloatingVoxel.x - currentSourcePositionFloatingVoxel.x;
    rayDirectorVectorFloatingVoxel.y = currentProjectionPositionFloatingVoxel.y - currentSourcePositionFloatingVoxel.y;
    rayDirectorVectorFloatingVoxel.z = currentProjectionPositionFloatingVoxel.z - currentSourcePositionFloatingVoxel.z;

    // Volume half length in floating voxel reference
    float3 volumeHalfLength;
    volumeHalfLength.x = static_cast<float>( p_volumeDimension.x ) / 2.F;
    volumeHalfLength.y = static_cast<float>( p_volumeDimension.y ) / 2.F;
    volumeHalfLength.z = static_cast<float>( p_volumeDimension.z ) / 2.F;

    float alphas[maxNumberOfDimensions];
    const auto raySize = FindProjectionAlphas_d( rayDirectorVectorFloatingVoxel, currentSourcePositionFloatingVoxel, p_volumeDimension, maxNumberOfDimensions, floatTolerance, alphas );

    // What are the alphas ?
    // this is the parameter which is used to identify a point on a line whose director vector is known.
    // if SP-> is the director vector of line (SP), for any point M on (SP), there is alpha, such that    SM-> = alpha x SP->
    // We compute those alphas for the current ray and for the intersection points with the underlying grids


#ifdef DEBUGPROJECTION
    if( globalProjectionIndex == debuggingIndex )
    {
        printf( "_______ sorted _______\n" );
        for( auto i = 0; i < totalAlphasSize; i++ )
        {
            printf( "   %f \n", alphas[i] );
        }
    }
#endif

    // projection performance
    auto total{ 0.F };
    auto totalWeight{ 0.F };
    for( auto i = 0; i < raySize - 1; i++ )
    {
        const auto centerAlpha = ( alphas[i + 1] + alphas[i] ) / 2.0;

        const auto pixX = static_cast<int>( volumeHalfLength.x + currentSourcePositionFloatingVoxel.x + centerAlpha * rayDirectorVectorFloatingVoxel.x );
        const auto pixY = static_cast<int>( volumeHalfLength.y + currentSourcePositionFloatingVoxel.y + centerAlpha * rayDirectorVectorFloatingVoxel.y );
        const auto pixZ = static_cast<int>( volumeHalfLength.z + currentSourcePositionFloatingVoxel.z + centerAlpha * rayDirectorVectorFloatingVoxel.z );

        const auto weight = ( alphas[i + 1] - alphas[i] );

        if( ( pixX >= 0 ) && ( pixX < p_volumeDimension.x ) && ( pixY >= 0 ) && ( pixY < p_volumeDimension.y )
            && ( pixZ >= 0 ) && ( pixZ < p_volumeDimension.z ) && weight > floatTolerance )
        {
            const auto currentIntersectionVoxelIndex = pixZ * p_volumeDimension.x * p_volumeDimension.y
                                                       + pixY * p_volumeDimension.x
                                                       + pixX;
            totalWeight += weight;
            total += weight * p_volumeBuffer[currentIntersectionVoxelIndex];
        }
    }
    if( totalWeight > floatTolerance )
    {
        p_projectionsBuffer[globalProjectionIndex] = total / totalWeight;
    }
}

// For this procedure, the coordinate system is the one centered in C, the pave volume
// center.
__global__ void CudaPerformBackProjection(
  const float * p_projectionsBuffer,
  const dim3 p_volumeDimension,
  const float3 p_volumeOriginInWorld,
  const float3 p_volumeVoxelsSpacing,
  float * p_volumeBuffer,
  const dim3 p_projectionsDimension,
  const float2 p_projectionsPixelSpacing,
  const float3 * p_projectionsOriginInWorld,
  const float3 * p_sourcesPositions )
{
    dim3 currentVoxel;
    currentVoxel.x = blockIdx.x * blockDim.x + threadIdx.x;
    currentVoxel.y = blockIdx.y * blockDim.y + threadIdx.y;
    currentVoxel.z = blockIdx.z * blockDim.z + threadIdx.z;

    if( currentVoxel.x < 0 || currentVoxel.x >= p_volumeDimension.x
        || currentVoxel.y < 0 || currentVoxel.y >= p_volumeDimension.y
        || currentVoxel.z < 0 || currentVoxel.z >= p_volumeDimension.z )
    {
        return;
    }

    const auto volumeSize{ p_volumeDimension.x * p_volumeDimension.y * p_volumeDimension.z };

    const auto volumeIndex = currentVoxel.z * p_volumeDimension.x * p_volumeDimension.y + currentVoxel.y * p_volumeDimension.x + currentVoxel.x;
    if( volumeIndex < 0 || volumeIndex >= volumeSize )
    {
#ifdef DEBUGPBACKROJECTION
        printf( "Id[%d] NOT POSSIBLE TO BE THERE !!! \n", volumeIndex );
#endif
        return;
    }


    const auto nbProjections = p_projectionsDimension.z;
    const auto nPixelsInOneRoi = p_projectionsDimension.x * p_projectionsDimension.y;
    const auto nProjectionsPixels = nbProjections * nPixelsInOneRoi;

    constexpr int maxNumberOfDimensions = 2000;
    constexpr auto floatTolerance = 0.000000000000000000000001F;

#ifdef DEBUGPBACKROJECTION
    // debugging index for some useful logging
    dim3 debuggingVoxel;
    debuggingVoxel.x = 329;
    debuggingVoxel.y = 494;
    debuggingVoxel.z = 20;
    const auto debuggingIndex = debuggingVoxel.z * p_volumeDimension.x * p_volumeDimension.y + debuggingVoxel.y * p_volumeDimension.x + debuggingVoxel.x;
    /*debuggingVoxel.x = 329;
    debuggingVoxel.y = 495;
    debuggingVoxel.z = 20;
    const auto debuggingIndex2 = debuggingVoxel.z * p_volumeDimension.x * p_volumeDimension.y + debuggingVoxel.y * p_volumeDimension.x + debuggingVoxel.x;*/
#endif

    p_volumeBuffer[volumeIndex] = 0.F;

    // coordinates in the FLOATING coordinate system (F-system) will all end with F
    // Origin in the F-system is the volume center.
    // unit is a voxelSpacing, dimension wise

    float3 currentVoxelF;
    currentVoxelF.x = static_cast<float>( currentVoxel.x ) - static_cast<float>( p_volumeDimension.x ) / 2.F;
    currentVoxelF.y = static_cast<float>( currentVoxel.y ) - static_cast<float>( p_volumeDimension.y ) / 2.F;
    currentVoxelF.z = static_cast<float>( currentVoxel.z ) - static_cast<float>( p_volumeDimension.z ) / 2.F;


#ifdef DEBUGPBACKROJECTION
    if( debuggingIndex == volumeIndex )
    {
        printf( "Id[%d]\t\t current voxel ( %d , %d , %d ) \n \t\t current voxel F ( %f , %f , %f ) \n", volumeIndex, currentVoxel.x, currentVoxel.y, currentVoxel.z, currentVoxelF.x, currentVoxelF.y, currentVoxelF.z );
    }
#endif

    auto total{ 0.F };
    auto totalWeight{ 0.F };
    for( auto projectionIndex{ 0 }; projectionIndex < nbProjections; projectionIndex++ )
    {
        float3 currentSourcePositionF;
        currentSourcePositionF.x = p_sourcesPositions[projectionIndex].x / p_volumeVoxelsSpacing.x;
        currentSourcePositionF.y = p_sourcesPositions[projectionIndex].y / p_volumeVoxelsSpacing.y;
        currentSourcePositionF.z = p_sourcesPositions[projectionIndex].z / p_volumeVoxelsSpacing.z;

        float3 currentProjectionsOriginF;
        currentProjectionsOriginF.x = p_projectionsOriginInWorld[projectionIndex].x / p_volumeVoxelsSpacing.x;
        currentProjectionsOriginF.y = p_projectionsOriginInWorld[projectionIndex].y / p_volumeVoxelsSpacing.y;
        currentProjectionsOriginF.z = p_projectionsOriginInWorld[projectionIndex].z / p_volumeVoxelsSpacing.z;

        float3 currentProjectionsEndF;
        currentProjectionsEndF.x = ( p_projectionsOriginInWorld[projectionIndex].x + static_cast<float>( p_projectionsDimension.x ) * p_projectionsPixelSpacing.x ) / p_volumeVoxelsSpacing.x;
        currentProjectionsEndF.y = ( p_projectionsOriginInWorld[projectionIndex].y + static_cast<float>( p_projectionsDimension.y ) * p_projectionsPixelSpacing.y ) / p_volumeVoxelsSpacing.y;
        currentProjectionsEndF.z = currentProjectionsOriginF.z;
         

        const auto pixelNeighborhoodSemiLength{ 5 }; // When the detected projection position is given by position (x,y), we look in a neighborhood of size 2*pixelNeighborhoodSemiLength+1
        bool constainsAtLeatOneProjectionPixel{ false };

        float3 currentProjectionPositionF; 
        currentProjectionPositionF.x = currentVoxelF.x + 0.5F;
        currentProjectionPositionF.y = currentVoxelF.y + 0.5F;
        currentProjectionPositionF.z = currentVoxelF.z + 0.5F;
        float3 currentProjectionRayDirectorVectorF;
        currentProjectionRayDirectorVectorF.x = currentProjectionPositionF.x - currentSourcePositionF.x;
        currentProjectionRayDirectorVectorF.y = currentProjectionPositionF.y - currentSourcePositionF.y;
        currentProjectionRayDirectorVectorF.z = currentProjectionPositionF.z - currentSourcePositionF.z;

        const auto alphaZ = fabs( currentProjectionRayDirectorVectorF.z ) > floatTolerance ? ( currentProjectionsOriginF.z - currentSourcePositionF.z ) / currentProjectionRayDirectorVectorF.z : 0.F;
        float2 intersectionF;
        intersectionF.x = currentSourcePositionF.x + currentProjectionRayDirectorVectorF.x * alphaZ;
        intersectionF.y = currentSourcePositionF.y + currentProjectionRayDirectorVectorF.y * alphaZ;
        if( intersectionF.x > currentProjectionsOriginF.x - static_cast<float>( pixelNeighborhoodSemiLength ) * p_projectionsPixelSpacing.x / p_volumeVoxelsSpacing.x 
            && intersectionF.x < currentProjectionsEndF.x  + static_cast<float>( pixelNeighborhoodSemiLength ) * p_projectionsPixelSpacing.x / p_volumeVoxelsSpacing.x 
            && intersectionF.y > currentProjectionsOriginF.y - static_cast<float>( pixelNeighborhoodSemiLength ) * p_projectionsPixelSpacing.y / p_volumeVoxelsSpacing.y 
            && intersectionF.y < currentProjectionsEndF.y + static_cast<float>( pixelNeighborhoodSemiLength ) * p_projectionsPixelSpacing.y / p_volumeVoxelsSpacing.y )    // we are on the detector :)
        { 
             constainsAtLeatOneProjectionPixel = true;
        }
        //         
        //    
        //        // For the eight corners of a voxel, we find the min and max in each dimension of the involved pixels on projection
//        float3 currentVoxelcurrentCornerF;
//        float3 currentRayDirectorVectorForCornerF;
//
//        float2 minProjectionPositionF;
//        minProjectionPositionF.x = currentProjectionsEndF.x + 1.F;
//        minProjectionPositionF.y = currentProjectionsEndF.y + 1.F;
//        float2 maxProjectionPositionF;
//        maxProjectionPositionF.x = currentProjectionsOriginF.x - 1.F;
//        maxProjectionPositionF.y = currentProjectionsOriginF.y - 1.F;
//        bool constainsAtLeatOneProjectionPixel{ false };
//        for( auto lagZ{ 0 }; lagZ <= 1; lagZ++ )
//        {
//            currentVoxelcurrentCornerF.z = currentVoxelF.z + static_cast<float>( lagZ ) - 0.5F;
//            currentRayDirectorVectorForCornerF.z = currentVoxelcurrentCornerF.z - currentSourcePositionF.z;
//            const auto alphaZ = fabs( currentRayDirectorVectorForCornerF.z ) > floatTolerance ? ( currentProjectionsOriginF.z - currentSourcePositionF.z ) / currentRayDirectorVectorForCornerF.z : 0.F;
//            float2 intersectionF;
//            for( auto lagX{ 0 }; lagX <= 1; lagX++ )
//            {
//                currentVoxelcurrentCornerF.x = currentVoxelF.x + static_cast<float>( lagX ) - 0.5F;
//                currentRayDirectorVectorForCornerF.x = currentVoxelcurrentCornerF.x - currentSourcePositionF.x;
//                intersectionF.x = currentSourcePositionF.x + currentRayDirectorVectorForCornerF.x * alphaZ;
//                for( auto lagY{ 0 }; lagY <= 1; lagY++ )
//                {
//                    currentVoxelcurrentCornerF.y = currentVoxelF.y + static_cast<float>( lagY ) - 0.5F;
//                    currentRayDirectorVectorForCornerF.y = currentVoxelcurrentCornerF.y - currentSourcePositionF.y;
//                    intersectionF.y = currentSourcePositionF.y + currentRayDirectorVectorForCornerF.y * alphaZ;
//
////#ifdef DEBUGPBACKROJECTION
////                    if( debuggingIndex == volumeIndex )
////                    {
////                        printf( "\t\t\t\t possible intersection PositionF ( %f , %f )\n",
////                                intersectionF.x,
////                                intersectionF.y );
////                    }
////#endif
//
//                    if( floatSupEq( intersectionF.x, currentProjectionsOriginF.x, floatTolerance )
//                        && intersectionF.x < currentProjectionsEndF.x
//                        && floatSupEq( intersectionF.y, currentProjectionsOriginF.y, floatTolerance )
//                        && intersectionF.y < currentProjectionsEndF.y )    // we are on the detector :)
//                    { 
//                        constainsAtLeatOneProjectionPixel = true;
//                        if( intersectionF.x < minProjectionPositionF.x )
//                        {
//                            minProjectionPositionF.x = intersectionF.x;
//                        }
//                        if( intersectionF.x > maxProjectionPositionF.x )
//                        {
//                            maxProjectionPositionF.x = intersectionF.x;
//                        }
//                        if( intersectionF.y < minProjectionPositionF.y )
//                        {
//                            minProjectionPositionF.y = intersectionF.y;
//                        }
//                        if( intersectionF.y > maxProjectionPositionF.y )
//                        {
//                            maxProjectionPositionF.y = intersectionF.y;
//                        }
//                    } 
//                }
//            }
//        }
//
//#ifdef DEBUGPBACKROJECTION
//        if( debuggingIndex == volumeIndex )
//        {
//            if( constainsAtLeatOneProjectionPixel )
//            {
//                printf( " \t\t we are inside detector \n" );
//            }
//            else
//            {
//                printf( " \t\t we are is outside detector \n" );
//            }
//            printf( " \t\t minProjectionPositionF ( %f , %f ) \n \t\t maxProjectionPositionF ( %f , %f ) \n",
//                    minProjectionPositionF.x,
//                    minProjectionPositionF.y,
//                    maxProjectionPositionF.x,
//                    maxProjectionPositionF.y );
//        }
//#endif

        if( constainsAtLeatOneProjectionPixel )
        { 
            //// min and max pixel computation:
            //// same process : by multiplying min(max)ProjectionPositionF by p_volumeVoxelsSpacing, we reset in World coordinates
            //// afterwards, by dividing by p_projectionsPixelSpacing (and casting in int), we can retrieve the pixel index
            //auto infProjectionPixelX = static_cast<int>( ( minProjectionPositionF.x  - currentProjectionsOriginF.x ) * p_volumeVoxelsSpacing.x / p_projectionsPixelSpacing.x );
            //infProjectionPixelX = infProjectionPixelX < 0 ? 0 : infProjectionPixelX >= p_projectionsDimension.x ? p_projectionsDimension.x - 1
            //                                                                                                    : infProjectionPixelX;    // clamping
            //auto infProjectionPixelY = static_cast<int>( ( minProjectionPositionF.y   - currentProjectionsOriginF.y ) * p_volumeVoxelsSpacing.y / p_projectionsPixelSpacing.y );
            //infProjectionPixelY = infProjectionPixelY < 0 ? 0 : infProjectionPixelY >= p_projectionsDimension.y ? p_projectionsDimension.y - 1
            //                                                                                                    : infProjectionPixelY;    // clamping
            //auto supProjectionPixelX = static_cast<int>( ( maxProjectionPositionF.x - currentProjectionsOriginF.x ) * p_volumeVoxelsSpacing.x / p_projectionsPixelSpacing.x );
            //supProjectionPixelX = supProjectionPixelX < 0 ? 0 : supProjectionPixelX >= p_projectionsDimension.x ? p_projectionsDimension.x - 1
            //                                                                                                    : supProjectionPixelX;    // clamping
            //auto supProjectionPixelY = static_cast<int>( ( maxProjectionPositionF.y  - currentProjectionsOriginF.y ) * p_volumeVoxelsSpacing.y / p_projectionsPixelSpacing.y );
            //supProjectionPixelY = supProjectionPixelY < 0 ? 0 : supProjectionPixelY >= p_projectionsDimension.y ? p_projectionsDimension.y - 1
            //                                                                                                    : supProjectionPixelY;    // clamping
//
//#ifdef DEBUGPBACKROJECTION
//            if( debuggingIndex == volumeIndex )
//            {
//                printf( " \t\t infProjectionPixel ( %d , %d ) \n \t\t supProjectionPixel ( %d , %d ) \n",
//                        infProjectionPixelX,
//                        infProjectionPixelY,
//                        supProjectionPixelX,
//                        supProjectionPixelY );
//            }
//#endif
             
            const auto pixelX = static_cast<int>( ( intersectionF.x - currentProjectionsOriginF.x ) * p_volumeVoxelsSpacing.x / p_projectionsPixelSpacing.x );
            const auto pixelY = static_cast<int>( ( intersectionF.y - currentProjectionsOriginF.y ) * p_volumeVoxelsSpacing.y / p_projectionsPixelSpacing.y );
            auto infProjectionPixelX = pixelX - pixelNeighborhoodSemiLength;
            if( infProjectionPixelX < 0 )
            {
                infProjectionPixelX = 0;
            }
            else if( infProjectionPixelX >= p_projectionsDimension.x )
            {
                infProjectionPixelX = p_projectionsDimension.x - 1;
            }
            
            auto supProjectionPixelX = pixelX + pixelNeighborhoodSemiLength;
            if( supProjectionPixelX < 0 )
            {
                supProjectionPixelX = 0;
            }
            else if( supProjectionPixelX >= p_projectionsDimension.x )
            {
                supProjectionPixelX = p_projectionsDimension.x - 1;
            }

            auto infProjectionPixelY = pixelY - pixelNeighborhoodSemiLength;
            if( infProjectionPixelY < 0 )
            {
                infProjectionPixelY = 0;
            }
            else if( infProjectionPixelY >= p_projectionsDimension.y )
            {
                infProjectionPixelY = p_projectionsDimension.y - 1;
            }
            auto supProjectionPixelY = pixelY + pixelNeighborhoodSemiLength;
            if( supProjectionPixelY < 0 )
            {
                supProjectionPixelY = 0;
            }
            else if( supProjectionPixelY >= p_projectionsDimension.y )
            {
                supProjectionPixelY = p_projectionsDimension.y - 1;
            }
            float3 alpha1;
            float3 alpha2;
            const auto currentIntersectionVoxelZlagIndex = projectionIndex * p_projectionsDimension.x * p_projectionsDimension.y;
            // Now we go through the invovled pixels
            currentProjectionPositionF.z = p_projectionsOriginInWorld[projectionIndex].z / p_volumeVoxelsSpacing.z;
            currentProjectionRayDirectorVectorF.z = currentProjectionPositionF.z - currentSourcePositionF.z;
            alpha1.z = fabs( currentProjectionRayDirectorVectorF.z ) > floatTolerance ? ( currentVoxelF.z - currentSourcePositionF.z ) / currentProjectionRayDirectorVectorF.z : 0.F;
            alpha2.z = fabs( currentProjectionRayDirectorVectorF.z ) > floatTolerance ? ( currentVoxelF.z + 1.F - currentSourcePositionF.z ) / currentProjectionRayDirectorVectorF.z : 0.F;

            for( auto projectionPixelY{ infProjectionPixelY }; projectionPixelY <= supProjectionPixelY; projectionPixelY++ )
            {
                const auto currentIntersectionVoxelYlagIndex = currentIntersectionVoxelZlagIndex + projectionPixelY * p_projectionsDimension.x ;
                currentProjectionPositionF.y = ( p_projectionsOriginInWorld[projectionIndex].y + ( static_cast<float>( projectionPixelY ) + 0.5F ) * p_projectionsPixelSpacing.y ) / p_volumeVoxelsSpacing.y;
                currentProjectionRayDirectorVectorF.y = currentProjectionPositionF.y - currentSourcePositionF.y;
                alpha1.y = fabs( currentProjectionRayDirectorVectorF.y ) > floatTolerance ? ( currentVoxelF.y - currentSourcePositionF.y ) / currentProjectionRayDirectorVectorF.y : 0.F;
                alpha2.y = fabs( currentProjectionRayDirectorVectorF.y ) > floatTolerance ? ( currentVoxelF.y + 1.F - currentSourcePositionF.y ) / currentProjectionRayDirectorVectorF.y : 0.F;

                for( auto projectionPixelX{ infProjectionPixelX }; projectionPixelX <= supProjectionPixelX; projectionPixelX++ )
                {
                    currentProjectionPositionF.x = ( p_projectionsOriginInWorld[projectionIndex].x + ( static_cast<float>( projectionPixelX ) + 0.5F ) * p_projectionsPixelSpacing.x ) / p_volumeVoxelsSpacing.x;
                    currentProjectionRayDirectorVectorF.x = currentProjectionPositionF.x - currentSourcePositionF.x;
                    alpha1.x = fabs( currentProjectionRayDirectorVectorF.x ) > floatTolerance ? ( currentVoxelF.x - currentSourcePositionF.x ) / currentProjectionRayDirectorVectorF.x : 0.F;
                    alpha2.x = fabs( currentProjectionRayDirectorVectorF.x ) > floatTolerance ? ( currentVoxelF.x + 1.F - currentSourcePositionF.x ) / currentProjectionRayDirectorVectorF.x : 0.F;

//                    
//#ifdef DEBUGPBACKROJECTION
//                    if( debuggingIndex == volumeIndex  )
//                    { 
//                        printf( " \t\t projectionPixel ( %d , %d ) of detector %d ",
//                                projectionPixelX,
//                                projectionPixelY,
//                                projectionIndex /*, 
//                                currentProjectionPositionF.x,
//                                currentProjectionPositionF.y,
//                                currentProjectionPositionF.z,
//                                currentProjectionRayDirectorVectorF.x,
//                                currentProjectionRayDirectorVectorF.y,
//                                currentProjectionRayDirectorVectorF.z,
//                                alpha1.x,
//                                alpha1.y,
//                                alpha1.z, 
//                                alpha2.x,
//                                alpha2.y,
//                                alpha2.z */
//                        );
//                    }
//#endif 

                    float3 alphaMins;    // alpha corresponding to the bottom left front vertex
                    alphaMins.x = fmin( alpha1.x, alpha2.x );
                    alphaMins.y = fmin( alpha1.y, alpha2.y );
                    alphaMins.z = fmin( alpha1.z, alpha2.z );
                    float3 alphaMaxs;    // alpha corresponding to the top right back vertex
                    alphaMaxs.x = fmax( alpha1.x, alpha2.x );
                    alphaMaxs.y = fmax( alpha1.y, alpha2.y );
                    alphaMaxs.z = fmax( alpha1.z, alpha2.z );


                    // We take the highest of the mins to catch the incoming intersection alpha
                    auto alphaMin = fmax( fmax( 0.F, alphaMins.x ), fmax( alphaMins.y, alphaMins.z ) );
                    // We take the lowest of the maxs to catch the outgoing intersection alpha
                    auto alphaMax = fmin( fmin( 1.F, alphaMaxs.x ), fmin( alphaMaxs.y, alphaMaxs.z ) );
                    // alphaMin and alphaMax constructions above guaranties the positiveness of 
                    if( floatSupEq( alphaMax, alphaMin ) )
                    {
                        const auto weight = ( alphaMax - alphaMin );

#ifdef DEBUGPBACKROJECTION
                        if( debuggingIndex == volumeIndex )
                        {
                            printf( "\t\t projectionPixel( %d, %d ) of detector %d (whose value is %f) has weight %f \n", 
                                    projectionPixelX,
                                    projectionPixelY,
                                    projectionIndex,
                                    p_projectionsBuffer[currentIntersectionVoxelYlagIndex + projectionPixelX],
                                    weight );
                        }
#endif 
                        if( weight > floatTolerance )
                        {
                            totalWeight += weight;
                            total += weight * p_projectionsBuffer[currentIntersectionVoxelYlagIndex + projectionPixelX];
                        }
                    }
#ifdef DEBUGPBACKROJECTION
                    //else
                    //{
                    //    if( debuggingIndex == volumeIndex   )
                    //    {
                    //        printf( " has no weight \n" );
                    //    }
                    //}
 
                    // //( 404 , 453 ) of detector 37 ( 405 , 452 ) of detector 37
                    //if( debuggingIndex == volumeIndex && projectionIndex == 37 )
                    //{  
                    //    if( ( projectionPixelX == 404 && projectionPixelY == 453 ) || ( projectionPixelX == 405 && projectionPixelY == 452 ) )
                    //        printf( " \t\t currentProjectionPositionF ( %f , %f , %f ) \n \t\t currentProjectionRayDirectorVectorF ( %f , %f , %f )\n \t\t alpha1 ( %f , %f , %f )\n \t\t alpha2 ( %f , %f , %f )\n \t\t alphaMins ( %f , %f , %f )\n \t\t alphaMaxs ( %f , %f , %f )\n \t\t alphaMin = %f  alphaMax = %f\n", 
                    //            currentProjectionPositionF.x,
                    //            currentProjectionPositionF.y,
                    //            currentProjectionPositionF.z,
                    //            currentProjectionRayDirectorVectorF.x,
                    //            currentProjectionRayDirectorVectorF.y,
                    //            currentProjectionRayDirectorVectorF.z,
                    //            alpha1.x,
                    //            alpha1.y,
                    //            alpha1.z, 
                    //            alpha2.x,
                    //            alpha2.y,
                    //            alpha2.z,
                    //            alphaMins.x,
                    //            alphaMins.y,
                    //            alphaMins.z,
                    //            alphaMaxs.x,
                    //            alphaMaxs.y,
                    //            alphaMaxs.z,
                    //            alphaMin,
                    //            alphaMax
                    //    );
                    //}
#endif
                }
            }
        } 
    }

    if( totalWeight > floatTolerance )
    {
        p_volumeBuffer[volumeIndex] = total / totalWeight;
    }

#ifdef DEBUGPBACKROJECTION
    if( debuggingIndex == volumeIndex )
    {
    printf( " \t\t result : %f \n ",
                p_volumeBuffer[volumeIndex] );
    }
#endif
}
void Projector::ExtractDimension( const vtkSmartPointer<vtkImageData> p_imageDataPtr, dim3 & p_dimension ) const
{
    if (p_imageDataPtr == nullptr)
    {
        return;
    }
    int dimensions[3];
    p_imageDataPtr->GetDimensions( dimensions ); 
    p_dimension.x = dimensions[0];
    p_dimension.y = dimensions[1];
    p_dimension.z = dimensions[2];
}
void Projector::ExtractSpacing( const vtkSmartPointer<vtkImageData> p_imageDataPtr, float3 & p_spacing ) const
{
    if( p_imageDataPtr == nullptr )
    {
        return;
    }
    double spacing[3];
    p_imageDataPtr->GetSpacing( spacing );
    p_spacing.x = static_cast<float>( spacing[0] );
    p_spacing.y = static_cast<float>( spacing[1] );
    p_spacing.z = static_cast<float>( spacing[2] );
}
void Projector::ExtractOrigin( const vtkSmartPointer<vtkImageData> p_imageDataPtr, float3 & p_origin ) const
{
    if( p_imageDataPtr == nullptr )
    {
        return;
    }
    double origin[3];
    p_imageDataPtr->GetOrigin( origin );
    p_origin.x = static_cast<float>( origin[0] );
    p_origin.y = static_cast<float>( origin[1] );
    p_origin.z = static_cast<float>( origin[2] );
}

void Projector::PerformProjection( const vtkSmartPointer<vtkImageData> p_volume, const Position3D & p_sourcePosition, vtkSmartPointer<vtkImageData> p_projectionsContainer ) const
{
    //// Device memory for volume elements 
    dim3 volumeDimensions; 
    this->ExtractDimension( p_volume, volumeDimensions );
    float3 volumeVoxelsSpacing;
    this->ExtractSpacing( p_volume, volumeVoxelsSpacing );
    float3 volumeOrigin;
    this->ExtractOrigin( p_volume, volumeOrigin );

    // declaration and preparation of buffer (for device)
    float * d_volumeBuffer;
    const auto volumeDimSize = volumeDimensions.z * volumeDimensions.y * volumeDimensions.x;
    unsigned int memSizeVolumeBuffer = volumeDimSize * sizeof( float );
    checkCudaErrors( hipMalloc( (void **)&d_volumeBuffer, memSizeVolumeBuffer ) );
    // copy to device memory
    const auto * volumeBuffer = static_cast<float *>( p_volume->GetScalarPointer() );
    checkCudaErrors( hipMemcpy( d_volumeBuffer, volumeBuffer, memSizeVolumeBuffer, hipMemcpyHostToDevice ) );

    //// Device memory for projections elements (result)
    dim3 projectionDimensions;
    this->ExtractDimension( p_projectionsContainer, projectionDimensions ); 
    
    float3 tempoProjectionsPixelSpacing;
    this->ExtractSpacing( p_projectionsContainer, tempoProjectionsPixelSpacing ); 
    float2 projectionsPixelSpacing;
    projectionsPixelSpacing.x = tempoProjectionsPixelSpacing.x;
    projectionsPixelSpacing.y = tempoProjectionsPixelSpacing.y;

    float * d_projectionBuffer;
    const auto projectionDimSize = projectionDimensions.z * projectionDimensions.y * projectionDimensions.x;
    unsigned int memSizeProjectionBuffer = projectionDimSize * sizeof( float );
    checkCudaErrors( hipMalloc( (void **)&d_projectionBuffer, memSizeProjectionBuffer ) );


    float3 * projectionsOriginInWorld = new float3[projectionDimensions.z];
    for( auto projectionIndex{ 0 }; projectionIndex < projectionDimensions.z; projectionIndex++ )
    {
        projectionsOriginInWorld[projectionIndex].x = m_tomoGeometry->projectionsRoisBottomLeftPositions().at( projectionIndex ).x;
        projectionsOriginInWorld[projectionIndex].y = m_tomoGeometry->projectionsRoisBottomLeftPositions().at( projectionIndex ).y;
        projectionsOriginInWorld[projectionIndex].z = m_tomoGeometry->detectorsZCommonPosition();
    }
    unsigned int memSizeProjectionOrigins = projectionDimensions.z * sizeof( float3 );
    float3 * d_projectionsOriginInWorld;
    checkCudaErrors( hipMalloc( (void **)&d_projectionsOriginInWorld, memSizeProjectionOrigins ) );
    checkCudaErrors( hipMemcpy( d_projectionsOriginInWorld, projectionsOriginInWorld, memSizeProjectionOrigins, hipMemcpyHostToDevice ) );

}


vtkSmartPointer<vtkImageData>  Projector::PerformProjection( vtkSmartPointer<vtkImageData>  p_volume ) const
{
    //// Device memory for volume elements
    dim3 volumeDimensions;
    volumeDimensions.x = m_tomoGeometry->GetVolume()->GetSize3D().x;
    volumeDimensions.y = m_tomoGeometry->GetVolume()->GetSize3D().y;
    volumeDimensions.z = m_tomoGeometry->GetVolume()->GetSize3D().z;

    float3 volumeOrigin;
    volumeOrigin.x = m_tomoGeometry->GetVolume()->GetBLF().x;
    volumeOrigin.y = m_tomoGeometry->GetVolume()->GetBLF().y;
    volumeOrigin.z = m_tomoGeometry->GetVolume()->GetBLF().z;

    float3 volumeVoxelsSpacing;
    volumeVoxelsSpacing.x = m_tomoGeometry->GetVolume()->GetVoxelSpacing().x;
    volumeVoxelsSpacing.y = m_tomoGeometry->GetVolume()->GetVoxelSpacing().y;
    volumeVoxelsSpacing.z = m_tomoGeometry->GetVolume()->GetVoxelSpacing().z;


    // declaration and preparation of buffer (for device)
    float * d_volumeBuffer;
    const auto volumeDimSize = volumeDimensions.z * volumeDimensions.y * volumeDimensions.x;
    unsigned int memSizeVolumeBuffer = volumeDimSize * sizeof( float );
    checkCudaErrors( hipMalloc( (void **)&d_volumeBuffer, memSizeVolumeBuffer ) );
    // copy to device memory
    const auto * volumeBuffer = static_cast<float *>( p_volume->GetScalarPointer() );
    checkCudaErrors( hipMemcpy( d_volumeBuffer, volumeBuffer, memSizeVolumeBuffer, hipMemcpyHostToDevice ) );

    //// Device memory for projections elements (result)
    dim3 projectionDimensions;
    projectionDimensions.z = m_tomoGeometry->nbProjectionsRois();
    projectionDimensions.x = m_tomoGeometry->projectionsRoisSize().x;
    projectionDimensions.y = m_tomoGeometry->projectionsRoisSize().y;

    float2 projectionsPixelSpacing;
    projectionsPixelSpacing.x = m_tomoGeometry->projectionsPixelSpacing().x;
    projectionsPixelSpacing.y = m_tomoGeometry->projectionsPixelSpacing().y;

    float * d_projectionBuffer;
    const auto projectionDimSize = projectionDimensions.z * projectionDimensions.y * projectionDimensions.x;
    unsigned int memSizeProjectionBuffer = projectionDimSize * sizeof( float );
    checkCudaErrors( hipMalloc( (void **)&d_projectionBuffer, memSizeProjectionBuffer ) );


    float3 * projectionsOriginInWorld = new float3[projectionDimensions.z];
    for( auto projectionIndex{ 0 }; projectionIndex < projectionDimensions.z; projectionIndex++ )
    {
        projectionsOriginInWorld[projectionIndex].x = m_tomoGeometry->projectionsRoisBottomLeftPositions().at( projectionIndex ).x;
        projectionsOriginInWorld[projectionIndex].y = m_tomoGeometry->projectionsRoisBottomLeftPositions().at( projectionIndex ).y;
        projectionsOriginInWorld[projectionIndex].z = m_tomoGeometry->detectorsZCommonPosition();
    }
    unsigned int memSizeProjectionOrigins = projectionDimensions.z * sizeof( float3 );
    float3 * d_projectionsOriginInWorld;
    checkCudaErrors( hipMalloc( (void **)&d_projectionsOriginInWorld, memSizeProjectionOrigins ) );
    checkCudaErrors( hipMemcpy( d_projectionsOriginInWorld, projectionsOriginInWorld, memSizeProjectionOrigins, hipMemcpyHostToDevice ) );

    //// Device memory for sources elements
    float3 * sourcePositionsInWorld = new float3[projectionDimensions.z];
    for( auto projectionIndex{ 0 }; projectionIndex < projectionDimensions.z; projectionIndex++ )
    {
        sourcePositionsInWorld[projectionIndex].x = m_tomoGeometry->sourcesXCommonPosition();
        sourcePositionsInWorld[projectionIndex].y = m_tomoGeometry->sourcesYPositions().at( projectionIndex );
        sourcePositionsInWorld[projectionIndex].z = m_tomoGeometry->sourcesZCommonPosition();
    }
    unsigned int memSizeSourcesPositions = projectionDimensions.z * sizeof( float3 );
    float3 * d_sourcePositionsInWorld;
    checkCudaErrors( hipMalloc( (void **)&d_sourcePositionsInWorld, memSizeSourcesPositions ) );
    checkCudaErrors( hipMemcpy( d_sourcePositionsInWorld, sourcePositionsInWorld, memSizeSourcesPositions, hipMemcpyHostToDevice ) );

    dim3 blockDims( 32, 16, 1 );
    dim3 gridDims( static_cast<unsigned int>( ceil( static_cast<double>( projectionDimensions.x ) / static_cast<double>( blockDims.x ) ) ),
                   static_cast<unsigned int>( ceil( static_cast<double>( projectionDimensions.y ) / static_cast<double>( blockDims.y ) ) ),
                   static_cast<unsigned int>( ceil( static_cast<double>( projectionDimensions.z ) / static_cast<double>( blockDims.z ) ) ) );

    //dim3 gridDims( 1,1,1 );
    std::cout << "projectionDimSize " << projectionDimSize << std::endl;
    std::cout << "projectionDimensions " << projectionDimensions.x << " , " << projectionDimensions.y << " , " << projectionDimensions.z << std::endl;
    std::cout << "blockDims " << blockDims.x << " , " << blockDims.y << " , " << blockDims.z << std::endl;
    std::cout << "gridDims " << gridDims.x << " , " << gridDims.y << " , " << gridDims.z << std::endl;


    CudaPerformProjection<<<gridDims, blockDims>>>( d_volumeBuffer, volumeDimensions, volumeOrigin, volumeVoxelsSpacing, d_projectionBuffer, projectionDimensions, projectionsPixelSpacing, d_projectionsOriginInWorld, d_sourcePositionsInWorld );

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    auto errorCode = hipGetLastError();
    std::cout << errorCode << std::endl;
    std::cout << hipGetErrorName( errorCode ) << std::endl;
    std::cout << hipGetErrorString( errorCode ) << std::endl;

    // Prepare data for output
    auto outputImage = vtkSmartPointer<vtkImageData> ::New();
    outputImage->SetDimensions( projectionDimensions.x, projectionDimensions.y, projectionDimensions.z );
    outputImage->SetSpacing( m_tomoGeometry->projectionsPixelSpacing().x, m_tomoGeometry->projectionsPixelSpacing().y, 1. );
    outputImage->AllocateScalars( VTK_FLOAT, 1 );
    auto finalImageBuffer = static_cast<float *>( outputImage->GetScalarPointer() );

    errorCode = hipMemcpy( finalImageBuffer, d_projectionBuffer, memSizeProjectionBuffer, hipMemcpyDeviceToHost );
    std::cout << errorCode << std::endl;
    std::cout << hipGetErrorName( errorCode ) << std::endl;
    std::cout << hipGetErrorString( errorCode ) << std::endl;

    auto tiffWriterOutput = vtkSmartPointer<vtkTIFFWriter>::New();
    tiffWriterOutput->SetFileName( "../Output.tiff" );
    tiffWriterOutput->SetInputData( outputImage );
    tiffWriterOutput->Write();

    checkCudaErrors( hipFree( d_sourcePositionsInWorld ) );
    free( sourcePositionsInWorld );
    checkCudaErrors( hipFree( d_projectionsOriginInWorld ) );
    free( projectionsOriginInWorld );
    checkCudaErrors( hipFree( d_projectionBuffer ) );
    checkCudaErrors( hipFree( d_volumeBuffer ) );

    return outputImage;

    ////double flopsPerMatrixMul = 2.0 * static_cast<double>( dimsA.x ) * static_cast<double>( dimsA.y ) * static_cast<double>( dimsB.x );
    ////double gigaFlops =
    ////  ( flopsPerMatrixMul * 1.0e-9f ) / ( msecPerMatrixMul / 1000.0f );
    ////printf(
    ////  "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
    ////  " WorkgroupSize= %u threads/block\n",
    ////  gigaFlops,
    ////  msecPerMatrixMul,
    ////  flopsPerMatrixMul,
    ////  threads.x * threads.y );
    //
    ////printf( "Checking computed result for correctness: " );
    ////bool correct = true;

    ////// test relative error by the formula
    //////     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    ////double eps = 1.e-6;    // machine zero

    ////for( int i = 0; i < static_cast<int>( dimsC.x * dimsC.y ); i++ )
    ////{
    ////    double abs_err = fabs( h_C[i] - ( dimsA.x * valB ) );
    ////    double dot_length = dimsA.x;
    ////    double abs_val = fabs( h_C[i] );
    ////    double rel_err = abs_err / abs_val / dot_length;

    ////    if( rel_err > eps )
    ////    {
    ////        printf( "Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x * valB, eps );
    ////        correct = false;
    ////    }
    ////}
    ////
    ////printf( "%s\n", correct ? "Result = PASS" : "Result = FAIL" );
    //
    ////printf(
    ////  "\nNOTE: The CUDA Samples are not meant for performance"
    ////  "measurements. Results may vary when GPU Boost is enabled.\n" );

    ////if( correct )
    ////{
    ////    return EXIT_SUCCESS;
    ////}
    ////else
    ////{
    ////    return EXIT_FAILURE;
    ////}
}

vtkSmartPointer<vtkImageData>  Projector::PerformBackProjection( vtkSmartPointer<vtkImageData>  p_projections ) const
{
    //// Device memory for volume elements (result)
    dim3 volumeDimensions;
    volumeDimensions.x = m_tomoGeometry->GetVolume()->GetSize3D().x;
    volumeDimensions.y = m_tomoGeometry->GetVolume()->GetSize3D().y;
    volumeDimensions.z = m_tomoGeometry->GetVolume()->GetSize3D().z;

    float3 volumeOrigin;
    volumeOrigin.x = m_tomoGeometry->GetVolume()->GetBLF().x;
    volumeOrigin.y = m_tomoGeometry->GetVolume()->GetBLF().y;
    volumeOrigin.z = m_tomoGeometry->GetVolume()->GetBLF().z;

    float3 volumeVoxelsSpacing;
    volumeVoxelsSpacing.x = m_tomoGeometry->GetVolume()->GetVoxelSpacing().x;
    volumeVoxelsSpacing.y = m_tomoGeometry->GetVolume()->GetVoxelSpacing().y;
    volumeVoxelsSpacing.z = m_tomoGeometry->GetVolume()->GetVoxelSpacing().z;

    float * d_volumeBuffer;
    const auto volumeDimSize = volumeDimensions.z * volumeDimensions.y * volumeDimensions.x;
    const auto memSizeVolumeBuffer = volumeDimSize * static_cast<unsigned int>( sizeof( float ) );
    checkCudaErrors( hipMalloc( (void **)&d_volumeBuffer, memSizeVolumeBuffer ) );


    //// Device memory for projections elements
    dim3 projectionDimensions;
    projectionDimensions.z = m_tomoGeometry->nbProjectionsRois();
    projectionDimensions.x = m_tomoGeometry->projectionsRoisSize().x;
    projectionDimensions.y = m_tomoGeometry->projectionsRoisSize().y;

    // declaration and preparation
    float * d_projectionBuffer;
    const auto projectionDimSize = projectionDimensions.z * projectionDimensions.y * projectionDimensions.x;
    unsigned int memSizeProjectionBuffer = projectionDimSize * sizeof( float );
    checkCudaErrors( hipMalloc( (void **)&d_projectionBuffer, memSizeProjectionBuffer ) );
    // copy to device memory
    auto projectionBuffer = static_cast<float *>( p_projections->GetScalarPointer() );
    checkCudaErrors( hipMemcpy( d_projectionBuffer, projectionBuffer, memSizeProjectionBuffer, hipMemcpyHostToDevice ) );


    float2 projectionsPixelsSpacing;
    projectionsPixelsSpacing.x = m_tomoGeometry->projectionsPixelSpacing().x;
    projectionsPixelsSpacing.y = m_tomoGeometry->projectionsPixelSpacing().y;

    float3 * projectionsOriginInWorld = new float3[projectionDimensions.z];
    for( auto projectionIndex{ 0 }; projectionIndex < projectionDimensions.z; projectionIndex++ )
    {
        projectionsOriginInWorld[projectionIndex].x = m_tomoGeometry->projectionsRoisBottomLeftPositions().at( projectionIndex ).x;
        projectionsOriginInWorld[projectionIndex].y = m_tomoGeometry->projectionsRoisBottomLeftPositions().at( projectionIndex ).y;
        projectionsOriginInWorld[projectionIndex].z = m_tomoGeometry->detectorsZCommonPosition();
    }
    unsigned int memSizeProjectionOrigins = projectionDimensions.z * sizeof( float3 );
    float3 * d_projectionsOriginInWorld;
    checkCudaErrors( hipMalloc( (void **)&d_projectionsOriginInWorld, memSizeProjectionOrigins ) );
    checkCudaErrors( hipMemcpy( d_projectionsOriginInWorld, projectionsOriginInWorld, memSizeProjectionOrigins, hipMemcpyHostToDevice ) );

    //// Device memory for sources elements
    float3 * sourcePositionsInWorld = new float3[projectionDimensions.z];
    for( auto projectionIndex{ 0 }; projectionIndex < projectionDimensions.z; projectionIndex++ )
    {
        sourcePositionsInWorld[projectionIndex].x = m_tomoGeometry->sourcesXCommonPosition();
        sourcePositionsInWorld[projectionIndex].y = m_tomoGeometry->sourcesYPositions().at( projectionIndex );
        sourcePositionsInWorld[projectionIndex].z = m_tomoGeometry->sourcesZCommonPosition();
    }
    unsigned int memSizeSourcesPositions = projectionDimensions.z * sizeof( float3 );
    float3 * d_sourcePositionsInWorld;
    checkCudaErrors( hipMalloc( (void **)&d_sourcePositionsInWorld, memSizeSourcesPositions ) );
    checkCudaErrors( hipMemcpy( d_sourcePositionsInWorld, sourcePositionsInWorld, memSizeSourcesPositions, hipMemcpyHostToDevice ) );

    dim3 blockDims( 32, 16, 1 );
    dim3 gridDims( static_cast<unsigned int>( ceil( static_cast<double>( volumeDimensions.x ) / static_cast<double>( blockDims.x ) ) ),
                   static_cast<unsigned int>( ceil( static_cast<double>( volumeDimensions.y ) / static_cast<double>( blockDims.y ) ) ),
                   static_cast<unsigned int>( ceil( static_cast<double>( volumeDimensions.z ) / static_cast<double>( blockDims.z ) ) ) );

    std::cout << "volumeDimensions " << volumeDimensions.x << " , " << volumeDimensions.y << " , " << volumeDimensions.z << std::endl;
    std::cout << "blockDims " << blockDims.x << " , " << blockDims.y << " , " << blockDims.z << std::endl;
    std::cout << "gridDims " << gridDims.x << " , " << gridDims.y << " , " << gridDims.z << std::endl;

    //const float * p_projectionsBuffer,
    //const dim3 p_volumeDimension,
    //const float3 p_volumeOriginInWorld,
    //const float3 p_volumeVoxelsSpacing,
    //float *p_volumeBuffer,
    //const dim3 p_projectionsDimension,
    //const float3 *p_projectionsOriginInWorld,
    //const float3 *p_sourcesPositions

    CudaPerformBackProjection<<<gridDims, blockDims>>>( d_projectionBuffer, volumeDimensions, volumeOrigin, volumeVoxelsSpacing, d_volumeBuffer, projectionDimensions, projectionsPixelsSpacing, d_projectionsOriginInWorld, d_sourcePositionsInWorld );

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    auto errorCode = hipGetLastError();
    std::cout << errorCode << std::endl;
    std::cout << hipGetErrorName( errorCode ) << std::endl;
    std::cout << hipGetErrorString( errorCode ) << std::endl;

    // Prepare data for output
    auto outputImage = vtkSmartPointer<vtkImageData> ::New();
    outputImage->SetDimensions( volumeDimensions.x, volumeDimensions.y, volumeDimensions.z );
    outputImage->SetSpacing( volumeVoxelsSpacing.x, volumeVoxelsSpacing.y, volumeVoxelsSpacing.z );
    outputImage->AllocateScalars( VTK_FLOAT, 1 );
    auto finalImageBuffer = static_cast<float *>( outputImage->GetScalarPointer() );

    errorCode = hipMemcpy( finalImageBuffer, d_volumeBuffer, memSizeVolumeBuffer, hipMemcpyDeviceToHost );
    std::cout << errorCode << std::endl;
    std::cout << hipGetErrorName( errorCode ) << std::endl;
    std::cout << hipGetErrorString( errorCode ) << std::endl;

    auto tiffWriterOutput = vtkSmartPointer<vtkTIFFWriter>::New();
    tiffWriterOutput->SetFileName( "../Output.tiff" );
    tiffWriterOutput->SetInputData( outputImage );
    tiffWriterOutput->Write();

    checkCudaErrors( hipFree( d_sourcePositionsInWorld ) );
    free( sourcePositionsInWorld );
    checkCudaErrors( hipFree( d_projectionsOriginInWorld ) );
    free( projectionsOriginInWorld );
    checkCudaErrors( hipFree( d_projectionBuffer ) );
    checkCudaErrors( hipFree( d_volumeBuffer ) );

    return outputImage;

    ////double flopsPerMatrixMul = 2.0 * static_cast<double>( dimsA.x ) * static_cast<double>( dimsA.y ) * static_cast<double>( dimsB.x );
    ////double gigaFlops =
    ////  ( flopsPerMatrixMul * 1.0e-9f ) / ( msecPerMatrixMul / 1000.0f );
    ////printf(
    ////  "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
    ////  " WorkgroupSize= %u threads/block\n",
    ////  gigaFlops,
    ////  msecPerMatrixMul,
    ////  flopsPerMatrixMul,
    ////  threads.x * threads.y );
    //
    ////printf( "Checking computed result for correctness: " );
    ////bool correct = true;

    ////// test relative error by the formula
    //////     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    ////double eps = 1.e-6;    // machine zero

    ////for( int i = 0; i < static_cast<int>( dimsC.x * dimsC.y ); i++ )
    ////{
    ////    double abs_err = fabs( h_C[i] - ( dimsA.x * valB ) );
    ////    double dot_length = dimsA.x;
    ////    double abs_val = fabs( h_C[i] );
    ////    double rel_err = abs_err / abs_val / dot_length;

    ////    if( rel_err > eps )
    ////    {
    ////        printf( "Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x * valB, eps );
    ////        correct = false;
    ////    }
    ////}
    ////
    ////printf( "%s\n", correct ? "Result = PASS" : "Result = FAIL" );
    //
    ////printf(
    ////  "\nNOTE: The CUDA Samples are not meant for performance"
    ////  "measurements. Results may vary when GPU Boost is enabled.\n" );

    ////if( correct )
    ////{
    ////    return EXIT_SUCCESS;
    ////}
    ////else
    ////{
    ////    return EXIT_FAILURE;
    ////}
}


/*


__global__ void blur( float * input_image, float * output_image, int width, int height )
{
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = ( offset - x ) / width;
    int fsize = 5;    // Filter size
    if( offset < width * height )
    {
        float output_val = 0.F;
        int hits = 0;
        for( int ox = -fsize; ox < fsize + 1; ++ox )
        {
            for( int oy = -fsize; oy < fsize + 1; ++oy )
            {
                if( ( x + ox ) > -1 && ( x + ox ) < width && ( y + oy ) > -1 && ( y + oy ) < height )
                {
                    const int currentoffset = ( offset + ox + oy * width );
                    output_val += input_image[currentoffset];
                    hits++;
                }
            }
        }
        output_image[offset] = output_val / static_cast<float>( hits );
    }
}

void filter( float * input_image, float * output_image, int width, int height )
{
    float * dev_input;
    float * dev_output;
    checkCudaErrors( hipMalloc( (void **)&dev_input, width * height *  sizeof( float ) ) );
    checkCudaErrors( hipMemcpy( dev_input, input_image, width * height * sizeof( float ), hipMemcpyHostToDevice ) );

    checkCudaErrors( hipMalloc( (void **)&dev_output, width * height * sizeof( float ) ) );

    dim3 blockDims( 512, 1, 1 );
    dim3 gridDims( (unsigned int)ceil( (double)( width * height / blockDims.x ) ), 1, 1 );

    blur<<<gridDims, blockDims>>>( dev_input, dev_output, width, height );
     
    checkCudaErrors( hipMemcpy( output_image, dev_output, width * height * sizeof( float ), hipMemcpyDeviceToHost ) );

    checkCudaErrors( hipFree( dev_input ) );
    checkCudaErrors( hipFree( dev_output ) );
}


ImageDataPtr CreateFloatTestImage( const int p_width, const int p_height, const int p_depth )
{
    auto imageTest = ImageDataPtr::New();

    imageTest->SetExtent( 0, p_width - 1, 0, p_height - 1, 0, p_depth - 1 );
    imageTest->SetOrigin( 0.0, 0.0, 0.0 );
    imageTest->SetSpacing( 0.140, 0.140, 1.0 );    // set the same spacing as the Viework detector (because why not)
    imageTest->AllocateScalars( VTK_FLOAT, 1 );
     
    auto value = 0.F;
    for( int depth = 0; depth < p_depth; depth++ )
    {
        for( int row = 0; row < p_height; row++ )
        {
            for( int col = 0; col < p_width; col++ )
            {
                auto pixel = static_cast<float *>( imageTest->GetScalarPointer( col, row, depth ) );
                pixel[0] = value;
                value = value + 1.F;
            }
        }
    }
    return imageTest;
}
  
*/